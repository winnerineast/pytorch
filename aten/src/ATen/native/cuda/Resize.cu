#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <ATen/native/cuda/Resize.cuh>

namespace at { namespace native {

Tensor& resize_cuda_(Tensor& self, IntArrayRef size) {
  auto* self_ = self.unsafeGetTensorImpl();
  resize_impl_cuda_(self_, size, /*strides=*/c10::nullopt);
  self_->maybe_zero_dim(size.size() == 0);
  return self;
}

Tensor& resize_as_cuda_(Tensor& self, const Tensor& the_template) {
  return resize_cuda_(self, the_template.sizes());
}

}}
