#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/core/op_registration/op_registration.h>
#include <cmath>

/* FakeQuantize Op for PerTensorAffine quantization scheme */
namespace at { namespace native {
namespace {
/* Fake-quantizes the 'inputs' tensor.
Args:
  X: Forward input tensor.
  scale: scale of per tensor affine quantization
  zero_point: zero_point of per tensor affine quantization
  quant_min: minimum quantized value
  quant_max: maximum quantized value
  quant_delay: Count of global steps for which to delay the quantization.
               See note below.
  iter: The current quantization iteration used for `quant_delay`.
Returns:
  Quantized tensor (double dtype).

Notes:
  - quant_delay might be set to non-zero to help weights stabilize in the
    beginning of the training.
  - quantization range [quant_min, quant_max]
*/
class FakeQuantizePerTensorAffineOp_forward : public c10::OperatorKernel {
 public:
  at::Tensor operator()(
      at::Tensor X,
      double scale,
      int64_t zero_point,
      int64_t quant_min = 0,
      int64_t quant_max = 255,
      int64_t quant_delay = 0,
      int64_t iter = 0
    ) {
    // Sanity checks.
    TORCH_CHECK(X.is_cuda());
    TORCH_CHECK(X.scalar_type() == ScalarType::Float);
    if (quant_min > quant_max) {
      throw std::invalid_argument("`quant_min` should be less than or equal to `quant_max`.");
    }
    if (zero_point < 0) {
      throw std::invalid_argument("`zero_point` must be a positive integer.");
    }
    if (quant_delay < 0) {
      throw std::invalid_argument("`quant_delay` must be a positive integer.");
    }

    if (quant_delay != 0 && iter < 0) {
      throw std::invalid_argument(
        "`iter` must be >=0 for non-zero `quant_delay`");
    }

    auto Y = at::empty_like(X);

    if (quant_delay > 0 && iter <= quant_delay) {
      Y.copy_(X);  // We might want to just return the input here.
      return Y;
    }

    float inv_scale = 1.0f / scale;
    at::cuda::CUDA_tensor_apply2<float, float>(
        X,
        Y,
        [=] __device__ (
            const float& input_val,
            float& result_val) {
          result_val = (fminf(quant_max, fmaxf(quant_min, (std::round(input_val * inv_scale + zero_point)))) - zero_point) * scale;
        });
    return Y;
  }
};

/* Backward path to fake-quantize the 'inputs' tensor.

Args:
  X: Forward input tensor.
  dY: Backward input tensor.
  scale: scale of per tensor affine quantization
  zero_point: zero_point of per tensor affine quantization
  quant_min: minimum quantized value
  quant_max: maximum quantized value
  quant_delay: Count of global steps for which to delay the quantization.
               See note in forward.
  iter: The current quantization iteration used for `quant_delay`.
Returns:
  Quantized tensor (double dtype).

Notes:
  - quant_delay might be set to non-zero to help weights stabilize in the
    beginning of the training.
  - quantization range [quant_min, quant_max]
*/
class FakeQuantizePerTensorAffineOp_backward : public c10::OperatorKernel {
 public:
  at::Tensor operator()(
      at::Tensor X,
      at::Tensor dY,
      double scale,
      int64_t zero_point,
      int64_t quant_min = 0,
      int64_t quant_max = 255,
      int64_t quant_delay = 0,
      int64_t iter = 0) {
    // Sanity checks.
    TORCH_CHECK(X.is_cuda());
    TORCH_CHECK(X.scalar_type() == ScalarType::Float);
    if (quant_min > quant_max) {
      throw std::invalid_argument("`quant_min` should be less than or equal to `quant_max`.");
    }
    if (zero_point < 0) {
      throw std::invalid_argument("`zero_point` must be a positive integer.");
    }
    if (quant_delay < 0) {
      throw std::invalid_argument("`quant_delay` must be a positive integer.");
    }
    if (X.numel() <= 0) {
      return X;
    }
    if (X.numel() != dY.numel()) {
      throw std::invalid_argument("`X` and `dY` are not the same size");
    }

    if (quant_delay != 0 && iter < 0) {
      throw std::invalid_argument(
        "`iter` must be >=0 for non-zero `quant_delay`");
    }

    auto dX = at::zeros_like(dY);
    if (quant_delay > 0 && iter <= quant_delay) {
      dX.copy_(dY);
      return dX;
    }

    float inv_scale = 1.0f / scale;
    auto mask = at::empty_like(dY);
    at::cuda::CUDA_tensor_apply2<float, float>(
        X,
        mask,
        [=] __device__ (
            const float& input_val,
            float& result_val) {
          float Xq = std::round(input_val * inv_scale + zero_point);
          result_val = float(Xq >= quant_min && Xq <= quant_max);
        });
    dX = mask * dY;
    return dX;
  }
};

static auto registry =
  c10::RegisterOperators()
  .op("quantized::fake_quantize_per_tensor_affine_forward(Tensor X, float scale, int zero_point, int quant_min = 0, int quant_max = 255, int quant_delay = 0, int iter = 0) -> Tensor",
      c10::RegisterOperators::options()
      .kernel<FakeQuantizePerTensorAffineOp_forward>(CUDATensorId()))
  .op("quantized::fake_quantize_per_tensor_affine_backward(Tensor X, Tensor dY, float scale, int zero_point, int quant_min = 0, int quant_max = 255, int quant_delay = 0, int iter = 0) -> Tensor",
      c10::RegisterOperators::options()
      .kernel<FakeQuantizePerTensorAffineOp_backward>(CUDATensorId()));

} // namespace
}} // namespace at::native
