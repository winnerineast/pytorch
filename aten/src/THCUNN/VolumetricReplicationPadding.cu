#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <THC/THCTensor.hpp>
#include <THCUNN/common.h>
#include <THC/THCDeviceTensor.cuh>
#include <THC/THCDeviceTensorUtils.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <THC/THCReduceApplyUtils.cuh>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCAtomics.cuh>
#include <THC/THCApply.cuh>

template <typename Dtype>
__global__ void VolumetricReplicationPadding_updateOutput(
  THCDeviceTensor<Dtype, 5> input,
  THCDeviceTensor<Dtype, 5> output,
  int pfront, int pback, int ptop, int pbottom, int pleft, int pright) {

  int outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;
  if (outputPointId >= (output.getSize(2) * output.getSize(3) *
                        output.getSize(4))) {
    return;
  }
  int outputPointX = outputPointId % output.getSize(4);
  int outputPointY = (outputPointId / output.getSize(4)) % output.getSize(3);
  int outputPointZ = outputPointId / (output.getSize(3) * output.getSize(4));

  int iStartX = max(0, -pleft);
  int iStartY = max(0, -ptop);
  int iStartZ = max(0, -pfront);
  int oStartX = max(0, pleft);
  int oStartY = max(0, ptop);
  int oStartZ = max(0, pfront);

  int inputPointX = min(max(pleft, outputPointX),
                        input.getSize(4) + pleft - 1) - oStartX + iStartX;
  int inputPointY = min(max(ptop, outputPointY),
                        input.getSize(3) + ptop - 1) - oStartY + iStartY;
  int inputPointZ = min(max(pfront, outputPointZ),
                        input.getSize(2) + pfront - 1) - oStartZ + iStartZ;

  Dtype valueToCopy =
      input[batch][plane][inputPointZ][inputPointY][inputPointX];
  output[batch][plane][outputPointZ][outputPointY][outputPointX] = valueToCopy;
}

template <typename Dtype>
__global__ void VolumetricReplicationPadding_updateGradInput(
  THCDeviceTensor<Dtype, 5> gradInput,
  THCDeviceTensor<Dtype, 5> gradOutput,
  int pfront, int pback, int ptop, int pbottom, int pleft, int pright) {
  int outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;

  if (outputPointId >= (gradOutput.getSize(2) * gradOutput.getSize(3) *
                        gradOutput.getSize(4))) {
    return;
  }
  int outputPointX = outputPointId % gradOutput.getSize(4);
  int outputPointY = (outputPointId / gradOutput.getSize(4)) %
      gradOutput.getSize(3);
  int outputPointZ = outputPointId / (gradOutput.getSize(3) *
      gradOutput.getSize(4));

  int iStartX = max(0, -pleft);
  int iStartY = max(0, -ptop);
  int iStartZ = max(0, -pfront);
  int oStartX = max(0, pleft);
  int oStartY = max(0, ptop);
  int oStartZ = max(0, pfront);

  int inputPointX = min(max(pleft, outputPointX),
                        gradInput.getSize(4) + pleft - 1) - oStartX + iStartX;
  int inputPointY = min(max(ptop, outputPointY),
                        gradInput.getSize(3) + ptop - 1) - oStartY + iStartY;
  int inputPointZ = min(max(pfront, outputPointZ),
                        gradInput.getSize(2) + pfront - 1) - oStartZ + iStartZ;

  Dtype valueToCopy =
      gradOutput[batch][plane][outputPointZ][outputPointY][outputPointX];
  atomicAdd(&gradInput[batch][plane][inputPointZ][inputPointY][inputPointX],
            valueToCopy);
}


#include <THCUNN/generic/VolumetricReplicationPadding.cu>
#include <THC/THCGenerateFloatTypes.h>
