#include <THCUNN/THCUNN.h>
#include <THC/THCTensor.hpp>
#include <THCUNN/common.h>
#include <ATen/native/cuda/im2col.cuh>

#include <TH/THHalf.h>
#include <THC/THCNumerics.cuh>
#include <THC/THCTensor.hpp>
#include <THC/THCStorage.hpp>

#include <THCUNN/generic/SpatialConvolutionLocal.cu>
#include <THC/THCGenerateFloatTypes.h>
